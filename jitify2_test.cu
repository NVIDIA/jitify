#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * * Redistributions of source code must retain the above copyright
 *   notice, this list of conditions and the following disclaimer.
 * * Redistributions in binary form must reproduce the above copyright
 *   notice, this list of conditions and the following disclaimer in the
 *   documentation and/or other materials provided with the distribution.
 * * Neither the name of NVIDIA CORPORATION nor the names of its
 *   contributors may be used to endorse or promote products derived
 *   from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#define JITIFY_ENABLE_EXCEPTIONS 1
#include "jitify2.hpp"

#include "example_headers/class_arg_kernel.cuh"
#include "example_headers/my_header1.cuh.jit"
#include "jitify2_test_kernels.cu.jit.hpp"

#include <chrono>
#include <iostream>
#include <string>
#include <vector>

#include "gtest/gtest.h"

#define CHECK_CUDA(call)                                                  \
  do {                                                                    \
    hipError_t status = call;                                               \
    if (status != hipSuccess) {                                         \
      const char* str;                                                    \
      cuda().GetErrorName()(status, &str);                                \
      std::cout << "(CUDA) returned " << str;                             \
      std::cout << " (" << __FILE__ << ":" << __LINE__ << ":" << __func__ \
                << "())" << std::endl;                                    \
      ASSERT_EQ(status, hipSuccess);                                    \
    }                                                                     \
  } while (0)

#define CHECK_CUDART(call)                                                \
  do {                                                                    \
    hipError_t status = call;                                            \
    if (status != hipSuccess) {                                          \
      std::cout << "(CUDART) returned " << hipGetErrorString(status);    \
      std::cout << " (" << __FILE__ << ":" << __LINE__ << ":" << __func__ \
                << "())" << std::endl;                                    \
      ASSERT_EQ(status, hipSuccess);                                     \
    }                                                                     \
  } while (0)

using namespace jitify2;
using namespace jitify2::reflection;

template <typename ValueType, typename ErrorType>
std::string get_error(
    const jitify2::detail::FallibleValue<ValueType, ErrorType>& x) {
  if (x) return "";
  return x.error();
}

void debug_print(const StringVec& v, const std::string& varname) {
  std::cerr << "--- BEGIN VECTOR " << varname << " ---\n";
  for (const auto& x : v) {
    std::cerr << x << "\n";
  }
  std::cerr << "--- END VECTOR " << varname << " ---" << std::endl;
}

bool contains(const StringVec& v, const std::string& s, const char* varname) {
  bool result = std::find(v.begin(), v.end(), s) != v.end();
  if (!result) debug_print(v, varname);
  return result;
}
bool not_contains(const StringVec& v, const std::string& s,
                  const char* varname) {
  bool result = std::find(v.begin(), v.end(), s) == v.end();
  if (!result) debug_print(v, varname);
  return result;
}

#define CONTAINS(src, target) contains(src, target, #src)
#define NOT_CONTAINS(src, target) not_contains(src, target, #src)

TEST(Jitify2Test, Simple) {
  static const char* const source = R"(
template <int N, typename T>
__global__ void my_kernel(T* data) {
  if (blockIdx.x != 0 || threadIdx.x != 0) return;
  T data0 = data[0];
  for( int i=0; i<N-1; ++i ) {
    data[0] *= data0;
  }
})";
  using dtype = float;
  dtype* d_data;
  CHECK_CUDART(hipMalloc((void**)&d_data, sizeof(dtype)));
  // Test serialization.
  auto program =
      Program::deserialize(Program("my_program", source)->serialize());
  ASSERT_EQ(get_error(program), "");
  auto preprog =
      PreprocessedProgram::deserialize(program->preprocess()->serialize());
  ASSERT_EQ(get_error(preprog), "");
  std::string kernel_inst =
      Template("my_kernel").instantiate(3, type_of(*d_data));
  auto compiled =
      CompiledProgram::deserialize(preprog->compile(kernel_inst)->serialize());
  ASSERT_EQ(get_error(compiled), "");
  auto linked = LinkedProgram::deserialize(compiled->link()->serialize());
  ASSERT_EQ(get_error(linked), "");

  // Test that kernel instantiation produces correct result.
  Kernel kernel = linked->load()->get_kernel(kernel_inst);
  dim3 grid(1), block(1);
  dtype h_data = 5;
  CHECK_CUDART(
      hipMemcpy(d_data, &h_data, sizeof(dtype), hipMemcpyHostToDevice));
  ASSERT_EQ(kernel->configure(grid, block)->launch(d_data), "");
  CHECK_CUDART(
      hipMemcpy(&h_data, d_data, sizeof(dtype), hipMemcpyDeviceToHost));
  EXPECT_FLOAT_EQ(h_data, 125.f);

  h_data = 5;
  CHECK_CUDART(
      hipMemcpy(d_data, &h_data, sizeof(dtype), hipMemcpyHostToDevice));
  ASSERT_EQ(kernel->configure_1d_max_occupancy()->launch(d_data), "");
  CHECK_CUDART(
      hipMemcpy(&h_data, d_data, sizeof(dtype), hipMemcpyDeviceToHost));
  EXPECT_FLOAT_EQ(h_data, 125.f);

  CHECK_CUDART(hipFree(d_data));
}

bool header_callback(const std::string& filename, std::string* source) {
  // On success, write to *source and return true, otherwise return false.
  if (filename == "example_headers/my_header4.cuh") {
    *source = R"(
#pragma once
template <typename T>
T pointless_func(T x) {
  return x;
};)";
    return true;
  } else {
    // Find this file through other mechanisms.
    return false;
  }
}

// Returns, e.g., "61" for a device of compute capability 6.1.
int get_current_device_arch() {
  int device;
  hipGetDevice(&device);
  int cc_major;
  hipDeviceGetAttribute(&cc_major, hipDeviceAttributeComputeCapabilityMajor, device);
  int cc_minor;
  hipDeviceGetAttribute(&cc_minor, hipDeviceAttributeComputeCapabilityMinor, device);
  int cc = cc_major * 10 + cc_minor;
  return cc;
}

TEST(Jitify2Test, MultipleKernels) {
  static const char* const source = R"(
#include "example_headers/my_header1.cuh"
#include "example_headers/my_header2.cuh"
#include "example_headers/my_header3.cuh"
#include "example_headers/my_header4.cuh"

__global__ void my_kernel1(const float* indata, float* outdata) {
  outdata[0] = indata[0] + 1;
  outdata[0] -= 1;
}

template <int C, typename T>
__global__ void my_kernel2(const float* indata, float* outdata) {
  for (int i = 0; i < C; ++i) {
    outdata[0] = pointless_func(identity(sqrt(square(negate(indata[0])))));
  }
})";

  enum { C = 123 };
  typedef float T;
  std::string kernel2_inst =
      Template("my_kernel2").instantiate<NonType<int, C>, T>();
  LoadedProgram program = Program("multiple_kernels_program", source)
                              ->preprocess({}, {}, header_callback)
                              ->load({"my_kernel1", kernel2_inst});
  ASSERT_EQ(get_error(program), "");

  T* indata;
  T* outdata;
  CHECK_CUDART(hipMalloc((void**)&indata, sizeof(T)));
  CHECK_CUDART(hipMalloc((void**)&outdata, sizeof(T)));
  T inval = 3.14159f;
  CHECK_CUDART(hipMemcpy(indata, &inval, sizeof(T), hipMemcpyHostToDevice));

  dim3 grid(1), block(1);
  ASSERT_EQ(program->get_kernel("my_kernel1")
                ->configure(grid, block)
                ->launch(indata, outdata),
            "");
  // These invocations are all equivalent.
  ASSERT_EQ(program->get_kernel(kernel2_inst)
                ->configure(grid, block)
                ->launch(indata, outdata),
            "");
  ASSERT_EQ(program
                ->get_kernel(Template("my_kernel2")
                                 .instantiate({reflect((int)C), reflect<T>()}))
                ->configure(grid, block)
                ->launch(indata, outdata),
            "");
  ASSERT_EQ(
      program->get_kernel(Template("my_kernel2").instantiate((int)C, Type<T>()))
          ->configure(grid, block)
          ->launch(indata, outdata),
      "");
  ASSERT_EQ(
      program
          ->get_kernel(
              Template("my_kernel2").instantiate((int)C, type_of(*indata)))
          ->configure(grid, block)
          ->launch(indata, outdata),
      "");
  ASSERT_EQ(
      program
          ->get_kernel(
              Template("my_kernel2").instantiate((int)C, instance_of(*indata)))
          ->configure(grid, block)
          ->launch(indata, outdata),
      "");

  T outval = 0;
  CHECK_CUDART(hipMemcpy(&outval, outdata, sizeof(T), hipMemcpyDeviceToHost));
  CHECK_CUDART(hipFree(outdata));
  CHECK_CUDART(hipFree(indata));

  EXPECT_FLOAT_EQ(inval, outval);
}

TEST(Jitify2Test, LaunchLatencyBenchmark) {
  static const char* const source = R"(
template <int N, int M, typename T, typename U>
__global__ void my_kernel(const T*, U*) {}
)";
  const size_t max_size = 2;
  // Note: It's faster (by ~300ns) to use custom keys, but we want to test
  // worst-case perf.
  ProgramCache<> cache(max_size, *Program("my_program", source)->preprocess(),
                       nullptr);
  float* idata = nullptr;
  uint8_t* odata = nullptr;
  dim3 grid(1), block(1);
  Kernel kernel = cache.get_kernel(
      Template("my_kernel")
          .instantiate(3, 4, type_of(*idata), type_of(*odata)));
  ASSERT_EQ(kernel->configure(grid, block)->launch(idata, odata), "");

  void* arg_ptrs[] = {&idata, &odata};

  int nrep = 10000;
  double dt_direct_ns = 1e99, dt_jitify_ns = 1e99;
  static const std::string kernel_inst =
      Template("my_kernel").instantiate(3, 4, type_of(*idata), type_of(*odata));
  for (int i = 0; i < nrep; ++i) {
    // Benchmark direct kernel launch.
    auto t0 = std::chrono::steady_clock::now();
    cuda().LaunchKernel()(kernel->function(), grid.x, grid.y, grid.z, block.x,
                          block.y, block.z, 0, 0, arg_ptrs, nullptr);
    auto dt = std::chrono::steady_clock::now() - t0;
    // Using the minimum is more robust than the average (though this test still
    // remains sensitive to the system environment and has been observed to fail
    // intermittently at a rate of <0.1%).
    dt_direct_ns = std::min(
        dt_direct_ns,
        (double)std::chrono::duration_cast<std::chrono::nanoseconds>(dt)
            .count());

    // Benchmark launch from cache.
    t0 = std::chrono::steady_clock::now();
    cache
        .get_kernel(
            // Note: It's faster to precompute this, but we want to test
            // worst-case perf.
            Template("my_kernel")
                .instantiate(3, 4, type_of(*idata), type_of(*odata)))
        ->configure(grid, block)
        ->launch(idata, odata);
    dt = std::chrono::steady_clock::now() - t0;
    dt_jitify_ns = std::min(
        dt_jitify_ns,
        (double)std::chrono::duration_cast<std::chrono::nanoseconds>(dt)
            .count());
  }
  double launch_time_direct_ns = dt_direct_ns;
  double launch_time_jitify_ns = dt_jitify_ns;
  // Ensure added latency is small.
  double tolerance_ns = 2500;  // 2.5us
  EXPECT_NEAR(launch_time_direct_ns, launch_time_jitify_ns, tolerance_ns);
}

class ScopeGuard {
  std::function<void()> func_;

 public:
  ScopeGuard(std::function<void()> func) : func_(std::move(func)) {}
  ~ScopeGuard() { func_(); }
  ScopeGuard(const ScopeGuard&) = delete;
  ScopeGuard& operator=(const ScopeGuard&) = delete;
  ScopeGuard(ScopeGuard&&) = delete;
  ScopeGuard& operator=(ScopeGuard&&) = delete;
};

inline bool remove_empty_dir(const char* path) {
#if defined(_WIN32) || defined(_WIN64)
  return ::_rmdir(path) == 0;
#else
  return ::rmdir(path) == 0;
#endif
}

TEST(Jitify2Test, ProgramCache) {
  static const char* const source = R"(
template <typename T>
__global__ void my_kernel(const T* __restrict__ idata, T* __restrict__ odata) {}
)";
  using key_type = uint32_t;
  size_t max_size = 2;
  static const char* const cache_path0 = "jitify2_test_cache";
  static const char* const cache_path = "jitify2_test_cache/subdir";
  ProgramCache<key_type> cache(max_size,
                               *Program("my_program", source)->preprocess(),
                               nullptr, cache_path);
  ScopeGuard scoped_cleanup_files([&] {
    cache.clear();
    remove_empty_dir(cache_path);
    remove_empty_dir(cache_path0);
  });

  auto check_hits = [&](size_t expected_hits, size_t expected_misses) {
    size_t num_hits, num_misses;
    cache.get_stats(&num_hits, &num_misses);
    EXPECT_EQ(num_hits, expected_hits);
    EXPECT_EQ(num_misses, expected_misses);
  };

  Kernel kernel;
  Template my_kernel("my_kernel");

  check_hits(0, 0);
  kernel = cache.get_kernel(/* key = */ 0, my_kernel.instantiate<float>());
  ASSERT_EQ(get_error(kernel), "");
  ASSERT_EQ(kernel->configure(1, 1)->launch(nullptr, nullptr), "");
  check_hits(0, 1);
  kernel = cache.get_kernel(/* key = */ 1, my_kernel.instantiate<double>());
  ASSERT_EQ(get_error(kernel), "");
  check_hits(0, 2);
  kernel = cache.get_kernel(/* key = */ 2, my_kernel.instantiate<int>());
  ASSERT_EQ(get_error(kernel), "");
  hipFunction_t function_int = kernel->function();
  check_hits(0, 3);
  cache.reset_stats();
  check_hits(0, 0);
  kernel = cache.get_kernel(/* key = */ 0, my_kernel.instantiate<float>());
  ASSERT_EQ(get_error(kernel), "");
  hipFunction_t function_float = kernel->function();
  check_hits(0, 1);
  kernel = cache.get_kernel(/* key = */ 2, my_kernel.instantiate<int>());
  ASSERT_EQ(get_error(kernel), "");
  EXPECT_EQ(kernel->function(), function_int);
  check_hits(1, 1);
  kernel = cache.get_kernel(/* key = */ 0, my_kernel.instantiate<float>());
  ASSERT_EQ(get_error(kernel), "");
  EXPECT_EQ(kernel->function(), function_float);
  check_hits(2, 1);
  LoadedProgram program =
      cache.get_program(/* key = */ 2, {my_kernel.instantiate<int>()});
  ASSERT_EQ(get_error(program), "");
  check_hits(3, 1);

  // Make sure cache dir was created.
  bool cache_path_is_dir;
  ASSERT_TRUE(jitify2::detail::path_exists(cache_path, &cache_path_is_dir));
  ASSERT_TRUE(cache_path_is_dir);
  // Make sure cache dir contains files.
  ASSERT_FALSE(remove_empty_dir(cache_path));
  // Now clear the cache.
  ASSERT_TRUE(cache.clear());
  EXPECT_EQ(cache.max_in_mem(), max_size);
  EXPECT_EQ(cache.max_files(), max_size);
  // Make sure cache dir still exists.
  ASSERT_TRUE(jitify2::detail::path_exists(cache_path, &cache_path_is_dir));
  ASSERT_TRUE(cache_path_is_dir);
  // Make sure cache dir is empty.
  ASSERT_TRUE(remove_empty_dir(cache_path));
  ASSERT_FALSE(jitify2::detail::path_exists(cache_path));

  max_size += 10;
  EXPECT_TRUE(cache.resize(max_size));
  EXPECT_EQ(cache.max_in_mem(), max_size);
  EXPECT_EQ(cache.max_files(), max_size);
  EXPECT_TRUE(cache.resize(max_size + 1, max_size + 2));
  EXPECT_EQ(cache.max_in_mem(), max_size + 1);
  EXPECT_EQ(cache.max_files(), max_size + 2);
}

TEST(Jitify2Test, ProgramCacheAutoKey) {
  static const char* const source = R"(
template <typename T>
__global__ void my_kernel(const T* __restrict__ idata, T* __restrict__ odata) {}
)";
  size_t max_size = 2;
  static const char* const cache_path0 = "jitify2_test_cache";
  static const char* const cache_path = "jitify2_test_cache/subdir";
  ProgramCache<> cache(max_size, *Program("my_program", source)->preprocess(),
                       nullptr, cache_path);
  ScopeGuard scoped_cleanup_files([&] {
    cache.clear();
    remove_empty_dir(cache_path);
    remove_empty_dir(cache_path0);
  });

  auto check_hits = [&](size_t expected_hits, size_t expected_misses) {
    size_t num_hits, num_misses;
    cache.get_stats(&num_hits, &num_misses);
    EXPECT_EQ(num_hits, expected_hits);
    EXPECT_EQ(num_misses, expected_misses);
  };

  Kernel kernel;
  Template my_kernel("my_kernel");

  check_hits(0, 0);
  kernel = cache.get_kernel(my_kernel.instantiate<float>());
  ASSERT_EQ(get_error(kernel), "");
  ASSERT_EQ(kernel->configure(1, 1)->launch(nullptr, nullptr), "");
  check_hits(0, 1);
  kernel = cache.get_kernel(my_kernel.instantiate<double>());
  ASSERT_EQ(get_error(kernel), "");
  check_hits(0, 2);
  kernel = cache.get_kernel(my_kernel.instantiate<int>());
  ASSERT_EQ(get_error(kernel), "");
  hipFunction_t function_int = kernel->function();
  check_hits(0, 3);
  cache.reset_stats();
  check_hits(0, 0);
  kernel = cache.get_kernel(my_kernel.instantiate<float>());
  ASSERT_EQ(get_error(kernel), "");
  hipFunction_t function_float = kernel->function();
  check_hits(0, 1);
  kernel = cache.get_kernel(my_kernel.instantiate<int>());
  ASSERT_EQ(get_error(kernel), "");
  EXPECT_EQ(kernel->function(), function_int);
  check_hits(1, 1);
  kernel = cache.get_kernel(my_kernel.instantiate<float>());
  ASSERT_EQ(get_error(kernel), "");
  EXPECT_EQ(kernel->function(), function_float);
  check_hits(2, 1);
  LoadedProgram program = cache.get_program({my_kernel.instantiate<int>()});
  ASSERT_EQ(get_error(program), "");
  check_hits(3, 1);

  // Make sure cache dir was created.
  bool cache_path_is_dir;
  ASSERT_TRUE(jitify2::detail::path_exists(cache_path, &cache_path_is_dir));
  ASSERT_TRUE(cache_path_is_dir);
  // Make sure cache dir contains files.
  ASSERT_FALSE(remove_empty_dir(cache_path));
  // Now clear the cache.
  ASSERT_TRUE(cache.clear());
  EXPECT_EQ(cache.max_in_mem(), max_size);
  EXPECT_EQ(cache.max_files(), max_size);
  // Make sure cache dir still exists.
  ASSERT_TRUE(jitify2::detail::path_exists(cache_path, &cache_path_is_dir));
  ASSERT_TRUE(cache_path_is_dir);
  // Make sure cache dir is empty.
  ASSERT_TRUE(remove_empty_dir(cache_path));
  ASSERT_FALSE(jitify2::detail::path_exists(cache_path));

  max_size += 10;
  EXPECT_TRUE(cache.resize(max_size));
  EXPECT_EQ(cache.max_in_mem(), max_size);
  EXPECT_EQ(cache.max_files(), max_size);
  EXPECT_TRUE(cache.resize(max_size + 1, max_size + 2));
  EXPECT_EQ(cache.max_in_mem(), max_size + 1);
  EXPECT_EQ(cache.max_files(), max_size + 2);
}

TEST(Jitify2Test, ProgramCacheFilenameSanitization) {
  static const char* const source = R"(__global__ void my_kernel() {})";
  const size_t max_size = 1;
  static const char* const cache_path = "jitify2_test_cache";
  // The filename is derived from the program name, so this checks that invalid
  // filename characters are automatically sanitized.
  ProgramCache<> cache(
      max_size, *Program("foo/bar/cat/dog\\:*?|<>", source)->preprocess(),
      nullptr, cache_path);
  ScopeGuard scoped_cleanup_files([&] {
    cache.clear();
    remove_empty_dir(cache_path);
  });
  *cache.get_kernel("my_kernel");
}

TEST(Jitify2Test, OfflinePreprocessing) {
  static const char* const extra_header_source = R"(
#pragma once
template <typename T>
T pointless_func(T x) {
  return x;
};)";
  size_t max_size = 10;
  // These variables come from the header generated by jitify_preprocess.
  ProgramCache<> cache(max_size, *jitify2_test_kernels_cu_jit,
                       jitify2_test_kernels_cu_headers_jit);
  enum { C = 123 };
  typedef float T;
  std::string kernel2_inst =
      Template("my_kernel2").instantiate<NonType<int, C>, T>();
  StringMap extra_headers = {{"my_header4.cuh", extra_header_source}};
  LoadedProgram program = cache.get_program(
      {"my_kernel1", kernel2_inst}, extra_headers, {"-include=my_header4.cuh"});
  ASSERT_EQ(get_error(program), "");

  T* indata;
  T* outdata;
  CHECK_CUDART(hipMalloc((void**)&indata, sizeof(T)));
  CHECK_CUDART(hipMalloc((void**)&outdata, sizeof(T)));
  T inval = 3.14159f;
  CHECK_CUDART(hipMemcpy(indata, &inval, sizeof(T), hipMemcpyHostToDevice));

  dim3 grid(1), block(1);
  ASSERT_EQ(program->get_kernel("my_kernel1")
                ->configure(grid, block)
                ->launch(indata, outdata),
            "");
  ASSERT_EQ(program->get_kernel(kernel2_inst)
                ->configure(grid, block)
                ->launch(indata, outdata),
            "");

  T outval = 0;
  CHECK_CUDART(hipMemcpy(&outval, outdata, sizeof(T), hipMemcpyDeviceToHost));
  CHECK_CUDART(hipFree(outdata));
  CHECK_CUDART(hipFree(indata));

  EXPECT_FLOAT_EQ(inval, outval);
}

TEST(Jitify2Test, Sha256) {
  EXPECT_EQ(jitify2::detail::sha256(""),
            "E3B0C44298FC1C149AFBF4C8996FB92427AE41E4649B934CA495991B7852B855");
  EXPECT_EQ(jitify2::detail::sha256(std::string(1, '\0')),
            "6E340B9CFFB37A989CA544E6BB780A2C78901D3FB33738768511A30617AFA01D");
  EXPECT_EQ(jitify2::detail::sha256("a"),
            "CA978112CA1BBDCAFAC231B39A23DC4DA786EFF8147C4E72B9807785AFEE48BB");
  EXPECT_EQ(jitify2::detail::sha256("abc"),
            "BA7816BF8F01CFEA414140DE5DAE2223B00361A396177A9CB410FF61F20015AD");
  EXPECT_EQ(
      jitify2::detail::sha256("The quick brown fox jumps over the lazy dog."),
      "EF537F25C895BFA782526529A9B63D97AA631564D5D789C2B765448C8635FB6C");
  EXPECT_EQ(
      jitify2::detail::sha256("The quick brown fox jumps over the lazy dog."
                              "The quick brown fox jumps over the lazy dog."
                              "The quick brown fox jumps over the lazy dog."
                              "The quick brown fox jumps over the lazy dog."),
      "F5EA20F5EDD6871D72D699C143C524BF9CEC13D06E9FA5763614EE3BA708C63E");
}

TEST(Jitify2Test, PathBase) {
  EXPECT_EQ(jitify2::detail::path_base("foo/bar/2"), "foo/bar");
  EXPECT_EQ(jitify2::detail::path_base("foo/bar/2/"), "foo/bar/2");
  EXPECT_EQ(jitify2::detail::path_base("foo"), "");
  EXPECT_EQ(jitify2::detail::path_base("/"), "");
#if defined _WIN32 || defined _WIN64
  EXPECT_EQ(jitify2::detail::path_base("foo\\bar\\2"), "foo\\bar");
  EXPECT_EQ(jitify2::detail::path_base("foo\\bar\\2\\"), "foo\\bar\\2");
  EXPECT_EQ(jitify2::detail::path_base("foo"), "");
  EXPECT_EQ(jitify2::detail::path_base("\\"), "");
#endif
}

TEST(Jitify2Test, PathJoin) {
  EXPECT_EQ(jitify2::detail::path_join("foo/bar", "2/1"), "foo/bar/2/1");
  EXPECT_EQ(jitify2::detail::path_join("foo/bar/", "2/1"), "foo/bar/2/1");
  EXPECT_EQ(jitify2::detail::path_join("foo/bar", "/2/1"), "");
#if defined _WIN32 || defined _WIN64
  EXPECT_EQ(jitify2::detail::path_join("foo\\bar", "2\\1"), "foo\\bar/2\\1");
  EXPECT_EQ(jitify2::detail::path_join("foo\\bar\\", "2\\1"), "foo\\bar\\2\\1");
  EXPECT_EQ(jitify2::detail::path_join("foo\\bar", "\\2\\1"), "");
#endif
}

TEST(Jitify2Test, PathSimplify) {
  EXPECT_EQ(jitify2::detail::path_simplify(""), "");
  EXPECT_EQ(jitify2::detail::path_simplify("/"), "/");
  EXPECT_EQ(jitify2::detail::path_simplify("//"), "/");
  EXPECT_EQ(jitify2::detail::path_simplify("/foo/bar"), "/foo/bar");
  EXPECT_EQ(jitify2::detail::path_simplify("foo/bar"), "foo/bar");
  EXPECT_EQ(jitify2::detail::path_simplify("/foo/./bar"), "/foo/bar");
  EXPECT_EQ(jitify2::detail::path_simplify("foo/./bar"), "foo/bar");
  EXPECT_EQ(jitify2::detail::path_simplify("/foo/../bar"), "/bar");
  EXPECT_EQ(jitify2::detail::path_simplify("foo/../bar"), "bar");
  EXPECT_EQ(jitify2::detail::path_simplify("/foo/cat/../../bar"), "/bar");
  EXPECT_EQ(jitify2::detail::path_simplify("foo/cat/../../bar"), "bar");
  EXPECT_EQ(jitify2::detail::path_simplify("/./bar"), "/bar");
  EXPECT_EQ(jitify2::detail::path_simplify("./bar"), "bar");
  EXPECT_EQ(jitify2::detail::path_simplify("../bar"), "../bar");
  EXPECT_EQ(jitify2::detail::path_simplify("../../bar"), "../../bar");
  EXPECT_EQ(jitify2::detail::path_simplify("../.././bar"), "../../bar");
  EXPECT_EQ(jitify2::detail::path_simplify(".././../bar"), "../../bar");
  EXPECT_EQ(jitify2::detail::path_simplify("./../../bar"), "../../bar");
  EXPECT_EQ(jitify2::detail::path_simplify("/foo/bar/.."), "/foo");
  EXPECT_EQ(jitify2::detail::path_simplify("foo/bar/.."), "foo");
  EXPECT_EQ(jitify2::detail::path_simplify("//foo///..////bar"), "/bar");
  EXPECT_EQ(jitify2::detail::path_simplify("foo/"), "foo/");
  EXPECT_EQ(jitify2::detail::path_simplify("/foo/"), "/foo/");
  EXPECT_EQ(jitify2::detail::path_simplify("foo/bar/"), "foo/bar/");
  EXPECT_EQ(jitify2::detail::path_simplify("/foo/bar/"), "/foo/bar/");
  EXPECT_EQ(jitify2::detail::path_simplify("foo/../bar/"), "bar/");
  EXPECT_EQ(jitify2::detail::path_simplify("/foo/../bar/"), "/bar/");
  EXPECT_EQ(jitify2::detail::path_simplify("/../foo"), "");  // Invalid path
  EXPECT_EQ(jitify2::detail::path_simplify("/foo/../../bar"), // Invalid path
            "");
  EXPECT_EQ(jitify2::detail::path_simplify("/.."), "");  // Invalid path
  EXPECT_EQ(jitify2::detail::path_simplify("/foo/../.."), "");  // Invalid path
#if defined _WIN32 || defined _WIN64
  EXPECT_EQ(jitify2::detail::path_simplify(R"(\)"), R"(\)");
  EXPECT_EQ(jitify2::detail::path_simplify(R"(\\)"), R"(\)");
  EXPECT_EQ(jitify2::detail::path_simplify(R"(\foo\bar)"), R"(\foo\bar)");
  EXPECT_EQ(jitify2::detail::path_simplify(R"(foo\bar)"), R"(foo\bar)");
  EXPECT_EQ(jitify2::detail::path_simplify(R"(\foo\.\bar)"), R"(\foo\bar)");
  EXPECT_EQ(jitify2::detail::path_simplify(R"(foo\.\bar)"), R"(foo\bar)");
  EXPECT_EQ(jitify2::detail::path_simplify(R"(\foo\..\bar)"), R"(\bar)");
  EXPECT_EQ(jitify2::detail::path_simplify(R"(foo\..\bar)"), R"(bar)");

  EXPECT_EQ(jitify2::detail::path_simplify(R"(\foo/.\bar)"),
            R"(\foo/bar)");
  EXPECT_EQ(jitify2::detail::path_simplify(R"(\foo/.\bar\./cat)"),
            R"(\foo/bar\cat)");
  EXPECT_EQ(jitify2::detail::path_simplify(R"(\foo/.\bar\../cat)"),
            R"(\foo/cat)");
#endif
}

TEST(Jitify2Test, Program) {
  static const char* const name = "my_program";
  static const char* const source = "/* empty source */";
  static const char* const header_name = "my_header";
  static const char* const header_source = "/* empty header */";
  Program program;
  ASSERT_EQ(static_cast<bool>(program), false);
  EXPECT_EQ(program.error(), "Uninitialized");
  EXPECT_THROW(*program, std::runtime_error);
  program = Program(name, source, {{header_name, header_source}});
  ASSERT_EQ(get_error(program), "");
  EXPECT_THROW(program.error(), std::runtime_error);
  EXPECT_EQ(program->name(), name);
  EXPECT_EQ(program->source(), source);
  EXPECT_EQ(program->header_sources().size(), size_t(1));
  ASSERT_EQ(program->header_sources().count(header_name), size_t(1));
  EXPECT_EQ(program->header_sources().at(header_name), header_source);
}

bool contains(const std::string& src, const std::string& target,
              const char* varname) {
  bool result = src.find(target) != std::string::npos;
  if (!result) {
    std::cerr << "--- BEGIN STRING " << varname << " ---\n"
              << src << "\n--- END STRING " << varname << " ---" << std::endl;
  }
  return result;
}

TEST(Jitify2Test, PreprocessedProgram) {
  // Tests source patching, header extraction, use of builtin headers, and basic
  // PreprocessedProgram API functionality.
  static const char* const name = "my_program";
  static const char* const source = R"(
#include <my_header1.cuh>
__global__ void my_kernel() {}
)";
  static const char* const header_name = "my_header1.cuh";
  Program program(name, source);
  ASSERT_EQ(get_error(program), "");
  PreprocessedProgram preprog = program->preprocess();
  ASSERT_EQ(static_cast<bool>(preprog), false);
  EXPECT_TRUE(CONTAINS(preprog.error(), "File not found"));
  preprog = program->preprocess({"-Iexample_headers"}, {"-lfoo"});
  ASSERT_EQ(get_error(preprog), "");
  EXPECT_EQ(preprog->name(), name);
  EXPECT_EQ(preprog->header_sources().count(header_name), size_t(1));
  EXPECT_TRUE(
      NOT_CONTAINS(preprog->remaining_compiler_options(), "-Iexample_headers"));
  EXPECT_EQ(preprog->remaining_linker_options(), StringVec({"-lfoo"}));
  EXPECT_NE(preprog->header_log(), "");
  EXPECT_EQ(preprog->compile_log(), "");
}

TEST(Jitify2Test, CompiledProgram) {
  // Tests compilation, lowered name lookup, and basic CompiledProgram API
  // functionality.
  static const char* const name = "my_program";
  static const char* const source = R"(
template <typename T>
__global__ void my_kernel() {}
)";
  static const char* const instantiation = "my_kernel<float>";
  static const char* const lowered_name = "_Z9my_kernelIfEvv";
  Program program(name, source);
  ASSERT_EQ(get_error(program), "");
  PreprocessedProgram preprog = program->preprocess();
  ASSERT_EQ(get_error(preprog), "");
  // TODO: Check that --remove-unused-globals is still needed.
  // Note: "--remove-unused-globals" is needed to WAR an issue in CUDA 12.0.
  CompiledProgram compiled = preprog->compile(
      instantiation, {}, {"--remove-unused-globals"}, {"-lfoo"});
  ASSERT_EQ(get_error(compiled), "");
  EXPECT_NE(compiled->ptx(), "");
  EXPECT_EQ(compiled->lowered_name_map().size(), size_t(1));
  ASSERT_EQ(compiled->lowered_name_map().count(instantiation), size_t(1));
  EXPECT_EQ(compiled->lowered_name_map().at(instantiation), lowered_name);
  std::unordered_multiset<std::string> linker_options;
  linker_options.insert(compiled->remaining_linker_options().begin(),
                        compiled->remaining_linker_options().end());
  EXPECT_EQ(linker_options.count("-lfoo"), 1);
  EXPECT_EQ(compiled->log(), "");
}

TEST(Jitify2Test, ConstantMemory) {
  static const char* const source = R"(
__constant__ int a;
__device__ int d;
namespace b { __constant__ int a; __device__ int d; }
namespace c { namespace b { __constant__ int a; __device__ int d; } }
namespace x { __constant__ int a = 3; __device__ int d = 7; }
namespace y { __constant__ int a[] = {4, 5}; __device__ int d[] = {8, 9}; }
namespace z { template <typename T> __constant__ T tv = 10; }

__global__ void constant_test(int* x) {
  x[0] = a;
  x[1] = b::a;
  x[2] = c::b::a;
  x[3] = d;
  x[4] = b::d;
  x[5] = c::b::d;
  x[6] = x::a;
  x[7] = x::d;
  x[8] = y::a[0];
  x[9] = y::a[1];
  x[10] = y::d[0];
  x[11] = y::d[1];
})";

  dim3 grid(1), block(1);
  {  // Test __constant__ look up in kernel using different namespaces.
    Kernel kernel = Program("constmem_program", source)
                        ->preprocess({"-std=c++14"})
                        // TODO: Use z::tv<float> in tests below.
                        ->get_kernel("constant_test", {"&z::tv<float>"});
    const LoadedProgramData& program = kernel->program();
    int dval;
    ASSERT_EQ(program.get_global_value("x::a", &dval), "");
    EXPECT_EQ(dval, 3);
    ASSERT_EQ(program.get_global_value("x::d", &dval), "");
    EXPECT_EQ(dval, 7);
    int darr[2];
    ASSERT_EQ(program.get_global_data("y::a", &darr[0], 2), "");
    EXPECT_EQ(darr[0], 4);
    EXPECT_EQ(darr[1], 5);
    ASSERT_EQ(program.get_global_value("y::d", &darr), "");
    EXPECT_EQ(darr[0], 8);
    EXPECT_EQ(darr[1], 9);
    int inval[] = {2, 4, 8, 12, 14, 18, 22, 26, 30, 34, 38, 42};
    constexpr int n_const = sizeof(inval) / sizeof(int);
    ASSERT_EQ(program.set_global_value("a", inval[0]), "");
    ASSERT_EQ(program.set_global_value("b::a", inval[1]), "");
    ASSERT_EQ(program.set_global_value("c::b::a", inval[2]), "");
    ASSERT_EQ(program.set_global_value("d", inval[3]), "");
    ASSERT_EQ(program.set_global_value("b::d", inval[4]), "");
    ASSERT_EQ(program.set_global_value("c::b::d", inval[5]), "");
    ASSERT_EQ(program.set_global_value("x::a", inval[6]), "");
    ASSERT_EQ(program.set_global_value("x::d", inval[7]), "");
    ASSERT_EQ(program.set_global_data("y::a", &inval[8], 2), "");
    int inarr[] = {inval[10], inval[11]};
    ASSERT_EQ(program.set_global_value("y::d", inarr), "");
    int* outdata;
    CHECK_CUDART(hipMalloc((void**)&outdata, n_const * sizeof(int)));
    ASSERT_EQ(kernel->configure(grid, block)->launch(outdata), "");
    CHECK_CUDART(hipDeviceSynchronize());
    int outval[n_const];
    CHECK_CUDART(
        hipMemcpy(outval, outdata, sizeof(outval), hipMemcpyDeviceToHost));
    for (int i = 0; i < n_const; i++) {
      EXPECT_EQ(inval[i], outval[i]);
    }
    CHECK_CUDART(hipFree(outdata));
  }
  {  // Test __constant__ array look up in header nested in both anonymous and
     // explicit namespace.
    static const char* const source2 =
        R"(#include "example_headers/constant_header.cuh")";
    Kernel kernel = Program("constmem_program2", source2)
                        ->preprocess()
                        ->get_kernel("constant_test2");
    const LoadedProgramData& program = kernel->program();
    int inval[] = {3, 5, 9, 13, 15, 19};
    constexpr int n_anon_const = sizeof(inval) / sizeof(int);
    std::string anon_prefix, anon_prefix2;
    if (jitify2::nvrtc().get_version() >= 11030) {
      // Internal linkage names changed in CUDA 11.3 (more robust mangling).
      anon_prefix = "constmem_program2::<unnamed>::";
      anon_prefix2 = "constmem_program2::(anonymous namespace)::";
    } else {
      anon_prefix = "<unnamed>::";
      anon_prefix2 = "(anonymous namespace)::";
    }
    ASSERT_EQ(program.set_global_data(anon_prefix + "b::a", inval, 3), "");
    ASSERT_EQ(program.set_global_data(anon_prefix + "b::d", inval + 3, 3), "");
    // Make sure alternative versions work too.
    ASSERT_EQ(program.set_global_data(anon_prefix2 + "b::a", inval, 3), "");
    ASSERT_EQ(program.set_global_data(anon_prefix2 + "b::d", inval + 3, 3), "");
    int* outdata;
    CHECK_CUDART(hipMalloc((void**)&outdata, n_anon_const * sizeof(int)));
    ASSERT_EQ(kernel->configure(grid, block)->launch(outdata), "");
    CHECK_CUDART(hipDeviceSynchronize());
    int outval[n_anon_const];
    CHECK_CUDART(
        hipMemcpy(outval, outdata, sizeof(outval), hipMemcpyDeviceToHost));
    for (int i = 0; i < n_anon_const; i++) {
      EXPECT_EQ(inval[i], outval[i]);
    }
    CHECK_CUDART(hipFree(outdata));
  }
}

TEST(Jitify2Test, InvalidPrograms) {
  // OK.
  EXPECT_EQ(get_error(Program("empty_program", "")->preprocess()), "");
  // OK.
  EXPECT_EQ(
      get_error(Program("found_header", "#include <cstdio>")->preprocess()),
      "");
  // Not OK.
  EXPECT_NE(
      get_error(
          Program("missing_header", "#include <cantfindme>")->preprocess()),
      "");
  // Not OK.
  EXPECT_NE(get_error(Program("bad_program", "NOT CUDA C!")->preprocess()), "");
}

TEST(Jitify2Test, CompileLTO_IR) {
  static const char* const source = R"(
const int arch = __CUDA_ARCH__ / 10;
)";

  if (!jitify2::nvrtc().GetNVVM()) return;  // Skip if not supported
  CompiledProgram program = Program("lto_nvvm_program", source)
                                ->preprocess({"-rdc=true", "-dlto"})
                                ->compile("", {}, {"-arch=compute_."});
  EXPECT_EQ(program->ptx().size(), 0);
  EXPECT_EQ(program->cubin().size(), 0);
  EXPECT_GT(program->nvvm().size(), 0);
  EXPECT_EQ(program->nvvm().size(), program->lto_ir().size());
  int current_arch = get_current_device_arch();
  LinkedProgram linked_program = program->link();
  if (CUDA_VERSION < 11040) {
    ASSERT_FALSE(linked_program.ok());
    ASSERT_TRUE(jitify2::detail::startswith(linked_program.error(),
                                            "Linking LTO IR is not supported"));
  } else {
    int arch;
    ASSERT_EQ(program->link()->load()->get_global_value("arch", &arch), "");
    EXPECT_EQ(arch, current_arch);
  }
}

TEST(Jitify2Test, LinkMultiplePrograms) {
  static const char* const source1 = R"(
__constant__ int c = 5;
__device__ int d = 7;
__device__ int f(int i) { return i + 11; }
)";

  static const char* const source2 = R"(
extern __constant__ int c;
extern __device__ int d;
extern __device__ int f(int);
__global__ void my_kernel(int* data) {
  *data = f(*data + c + d);
}
)";

  CompiledProgram program1 = Program("linktest_program1", source1)
                                 ->preprocess({"-rdc=true"})
                                 ->compile();
  CompiledProgram program2 = Program("linktest_program2", source2)
                                 ->preprocess({"-rdc=true"})
                                 ->compile("my_kernel");
  // TODO: Consider allowing refs not ptrs for programs, and also addding a
  //         get_kernel() shortcut method to LinkedProgram.
  Kernel kernel = LinkedProgram::link({&program1, &program2})
                      ->load()
                      ->get_kernel("my_kernel");
  int* d_data;
  CHECK_CUDART(hipMalloc((void**)&d_data, sizeof(int)));
  int h_data = 3;
  CHECK_CUDART(
      hipMemcpy(d_data, &h_data, sizeof(int), hipMemcpyHostToDevice));
  ASSERT_EQ(kernel->configure(1, 1)->launch(d_data), "");
  CHECK_CUDART(
      hipMemcpy(&h_data, d_data, sizeof(int), hipMemcpyDeviceToHost));
  EXPECT_EQ(h_data, 26);
  CHECK_CUDART(hipFree(d_data));
}

TEST(Jitify2Test, LinkLTO) {
  static const char* const source1 = R"(
__constant__ int c = 5;
__device__ int d = 7;
extern "C"
__device__ int f(int i) { return i + 11; }
)";

  static const char* const source2 = R"(
extern __constant__ int c;
extern __device__ int d;
extern "C" __device__ int f(int);
__global__ void my_kernel(int* data) {
  *data = f(*data + c + d);
}
)";

  if (!jitify2::nvrtc().GetNVVM()) return;  // Skip if not supported

  // **TODO: Work out what code-type mixing is allowed when linking.
  CompiledProgram program1 = Program("linktest_program1", source1)
                                 ->preprocess({"-rdc=true", "-dlto"})
                                 ->compile("");
  CompiledProgram program2 = Program("linktest_program2", source2)
                                 ->preprocess({"-rdc=true", "-dlto"})
                                 ->compile("my_kernel");
  // TODO: Consider allowing refs not ptrs for programs, and also addding a
  //         get_kernel() shortcut method to LinkedProgram.
  LinkedProgram linked_program = LinkedProgram::link({&program1, &program2});
  if (CUDA_VERSION < 11040) {
    ASSERT_FALSE(linked_program.ok());
    ASSERT_TRUE(jitify2::detail::startswith(linked_program.error(),
                                            "Linking LTO IR is not supported"));
    return;
  }
  Kernel kernel = linked_program->load()->get_kernel("my_kernel");
  int* d_data;
  CHECK_CUDART(hipMalloc((void**)&d_data, sizeof(int)));
  int h_data = 3;
  CHECK_CUDART(
      hipMemcpy(d_data, &h_data, sizeof(int), hipMemcpyHostToDevice));
  ASSERT_EQ(kernel->configure(1, 1)->launch(d_data), "");
  CHECK_CUDART(
      hipMemcpy(&h_data, d_data, sizeof(int), hipMemcpyDeviceToHost));
  EXPECT_EQ(h_data, 26);
  CHECK_CUDART(hipFree(d_data));
}

TEST(Jitify2Test, LinkExternalFiles) {
  static const char* const source1 = R"(
__constant__ int c = 5;
__device__ int d = 7;
__device__ int f(int i) { return i + 11; })";

  static const char* const source2 = R"(
extern __constant__ int c;
extern __device__ int d;
extern __device__ int f(int);
__global__ void my_kernel(int* data) {
  *data = f(*data + c + d);
})";

  // Ensure temporary file is deleted at the end.
  std::unique_ptr<const char, int (*)(const char*)> ptx_filename(
      "example_headers/linktest.ptx", std::remove);
  {
    std::ofstream ptx_file(ptx_filename.get());
    ptx_file.exceptions(std::ofstream::failbit | std::ofstream::badbit);
    ptx_file << Program("linktest_program1", source1)
                    ->preprocess({"-rdc=true"})
                    ->compile()
                    ->ptx();
  }
  const std::vector<std::string> linker_options0 = {"-Lexample_headers",
                                                    "-llinktest.ptx"};
  for (bool use_culink : {false, true}) {
    std::vector<std::string> linker_options = linker_options0;
    if (use_culink) {
      linker_options.push_back("--use-culink");
    }
    Kernel kernel = Program("linktest_program2", source2)
                        ->preprocess({"-rdc=true"}, linker_options)
                        ->get_kernel("my_kernel");
    int* d_data;
    CHECK_CUDART(hipMalloc((void**)&d_data, sizeof(int)));
    int h_data = 3;
    CHECK_CUDART(
        hipMemcpy(d_data, &h_data, sizeof(int), hipMemcpyHostToDevice));
    ASSERT_EQ(kernel->configure(1, 1)->launch(d_data), "");
    CHECK_CUDART(
        hipMemcpy(&h_data, d_data, sizeof(int), hipMemcpyDeviceToHost));
    EXPECT_EQ(h_data, 26);
    CHECK_CUDART(hipFree(d_data));
  }
}

namespace a {
__host__ __device__ int external_device_func(int i) { return i + 1; }
}  // namespace a

TEST(Jitify2Test, LinkCurrentExecutable) {
  static const char* const source = R"(
namespace a {
extern __device__ int external_device_func(int);
}
__global__ void my_kernel(int* data) {
  *data = a::external_device_func(*data);
})";
  Kernel kernel = Program("selflink_program", source)
                      ->preprocess({"-rdc=true"}, {"-l."})
                      ->get_kernel("my_kernel");
  int* d_data;
  CHECK_CUDART(hipMalloc((void**)&d_data, sizeof(int)));
  int h_data = 3;
  CHECK_CUDART(
      hipMemcpy(d_data, &h_data, sizeof(int), hipMemcpyHostToDevice));
  ASSERT_EQ(kernel->configure(1, 1)->launch(d_data), "");
  CHECK_CUDART(
      hipMemcpy(&h_data, d_data, sizeof(int), hipMemcpyDeviceToHost));
  EXPECT_EQ(h_data, 4);
  CHECK_CUDART(hipFree(d_data));
}

TEST(Jitify2Test, ClassKernelArg) {
  static const char* const source = R"(
#include "example_headers/class_arg_kernel.cuh"
)";

  int h_data;
  int* d_data;
  CHECK_CUDART(hipMalloc((void**)&d_data, sizeof(int)));

  PreprocessedProgram preprog =
      Program("class_kernel_arg_program", source)->preprocess();
  ConfiguredKernel configured_kernel =
      preprog->get_kernel(Template("class_arg_kernel").instantiate<Arg>())
          ->configure(1, 1);

  {  // Test that we can pass an arg object to a kernel.
    Arg arg(-1);
    ASSERT_EQ(configured_kernel->launch(d_data, arg), "");
    CHECK_CUDART(hipDeviceSynchronize());
    CHECK_CUDART(
        hipMemcpy(&h_data, d_data, sizeof(int), hipMemcpyDeviceToHost));
    EXPECT_EQ(arg.x, h_data);
  }

  {  // Test that we can pass an arg object rvalue to a kernel.
    int value = -2;
    ASSERT_EQ(configured_kernel->launch(d_data, Arg(value)), "");
    CHECK_CUDART(hipDeviceSynchronize());
    CHECK_CUDART(
        hipMemcpy(&h_data, d_data, sizeof(int), hipMemcpyDeviceToHost));
    EXPECT_EQ(value, h_data);
  }

  {  // Test that we can pass an arg object reference to a kernel.
    std::unique_ptr<Arg> arg(new Arg(-3));
    // References are passed as pointers since refernces are just pointers from
    // an ABI point of view.
    ASSERT_EQ(
        preprog->get_kernel(Template("class_arg_ref_kernel").instantiate<Arg>())
            ->configure(1, 1)
            ->launch(d_data, arg.get()),
        "");
    CHECK_CUDART(
        hipMemcpy(&h_data, d_data, sizeof(int), hipMemcpyDeviceToHost));
    EXPECT_EQ(arg->x, h_data);
  }

  {  // Test that we can pass an arg object reference to a kernel
    std::unique_ptr<Arg> arg(new Arg(-4));
    ASSERT_EQ(
        preprog->get_kernel(Template("class_arg_ptr_kernel").instantiate<Arg>())
            ->configure(1, 1)
            ->launch(d_data, arg.get()),
        "");
    CHECK_CUDART(
        hipMemcpy(&h_data, d_data, sizeof(int), hipMemcpyDeviceToHost));
    EXPECT_EQ(arg->x, h_data);
  }

  CHECK_CUDART(hipFree(d_data));
}

TEST(Jitify2Test, GetAttribute) {
  static const char* const source = R"(
__global__ void get_attribute_kernel(int* out, const int* in) {
  __shared__ int buffer[4096];
  buffer[threadIdx.x] = in[threadIdx.x];
  __syncthreads();
  out[threadIdx.y] = buffer[threadIdx.x];
}
)";

  // Checks that we can get function attributes.
  int attrval;
  ASSERT_EQ(Program("get_attribute_program", source)
                ->preprocess()
                ->get_kernel("get_attribute_kernel")
                ->get_attribute(HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, &attrval),
            "");
  EXPECT_EQ(attrval, 4096 * (int)sizeof(int));
}

TEST(Jitify2Test, SetAttribute) {
  static const char* const source = R"(
__global__ void set_attribute_kernel(int* out, int* in) {
  extern __shared__ int buffer[];
  buffer[threadIdx.x] = in[threadIdx.x];
  __syncthreads();
  out[threadIdx.y] = buffer[threadIdx.x];
}
)";

  int* in;
  CHECK_CUDART(hipMalloc((void**)&in, sizeof(int)));
  int* out;
  CHECK_CUDART(hipMalloc((void**)&out, sizeof(int)));

  // Query the maximum supported shared bytes per block.
  hipDevice_t device;
  CHECK_CUDA(cuda().DeviceGet()(&device, 0));
  int shared_bytes;
  CHECK_CUDA(cuda().DeviceGetAttribute()(
      &shared_bytes, hipDeviceAttributeSharedMemPerBlockOptin,
      device));

  Kernel kernel = Program("set_attribute_program", source)
                      ->preprocess()
                      ->get_kernel("set_attribute_kernel");
  ASSERT_EQ(kernel->set_attribute(
                HIP_FUNC_ATTRIBUTE_MAX_DYNAMIC_SHARED_SIZE_BYTES, shared_bytes),
            "");

  dim3 grid(1), block(1);
  // This kernel will fail on Volta+ unless the set attribute succeeded.
  ASSERT_EQ(kernel->configure(grid, block, shared_bytes)->launch(out, in), "");

  CHECK_CUDART(hipFree(out));
  CHECK_CUDART(hipFree(in));
}

TEST(Jitify2Test, RemoveUnusedGlobals) {
  static const char* const source = R"(
struct Foo { static const int value = 7; };
struct Bar { int a; double b; };
__device__ float used_scalar;
__device__ float used_array[2];
__device__ Bar used_struct;
__device__ int used_scalar_init = 3;
__device__ int used_array_init[] = {4, 5};
__device__ Bar used_struct_init = {6, 0.0};
__device__ float unused_scalar;
__device__ float unused_array[3];
__device__ Bar unused_struct;
__device__ int unused_scalar_init = 3;
__device__ int unused_array_init[] = {4, 5};
__device__ Bar unused_struct_init = {6, 0.0};
__device__ float reg, ret, bra;  // Tricky name
__global__ void foo_kernel(int* data) {
  if (blockIdx.x != 0 || threadIdx.x != 0) return;
  used_scalar = 1.f;
  used_array[1] = 2.f;
  used_struct.b = 3.f;
  used_scalar_init = 1;
  used_array_init[1] = 2;
  used_struct_init.b = 3.f;
  __syncthreads();
  *data += Foo::value + used_scalar + used_array[1] + used_struct.b;
  // printf produces global symbols named $str.
  printf("printf test: *data = %i\n", *data);
})";
  CompiledProgram compiled =
      Program("unused_globals_source", source)
          ->preprocess(
              // Note: Flag added twice to test handling of repeats.
              {"-remove-unused-globals", "--remove-unused-globals"})
          ->compile("foo_kernel");
  const std::string& ptx = compiled->ptx();
  EXPECT_TRUE(ptx.find(".global .align 4 .f32 used_scalar;") !=
              std::string::npos);
  // Note: PTX represents arrays and structs as .b8 instead of the actual type.
  EXPECT_TRUE(ptx.find(".global .align 4 .b8 used_array[8];") !=
              std::string::npos);
  EXPECT_TRUE(ptx.find(".global .align 8 .b8 used_struct[16];") !=
              std::string::npos);
  EXPECT_TRUE(ptx.find(".global .align 4 .u32 used_scalar_init = 3;") !=
              std::string::npos);
  EXPECT_TRUE(ptx.find(".global .align 4 .b8 used_array_init[8] = {4, 0, 0, 0, "
                       "5, 0, 0, 0};") != std::string::npos);
  EXPECT_TRUE(ptx.find(".global .align 8 .b8 used_struct_init[16] = {6, 0, 0, "
                       "0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};") !=
              std::string::npos);
  EXPECT_FALSE(ptx.find("_ZN3Foo5valueE") != std::string::npos);
  EXPECT_FALSE(ptx.find("unused_scalar;") != std::string::npos);
  EXPECT_FALSE(ptx.find("unused_array;") != std::string::npos);
  EXPECT_FALSE(ptx.find("unused_struct;") != std::string::npos);
  EXPECT_FALSE(ptx.find("unused_scalar_init;") != std::string::npos);
  EXPECT_FALSE(ptx.find("unused_array_init;") != std::string::npos);
  EXPECT_FALSE(ptx.find("unused_struct_init;") != std::string::npos);
  EXPECT_FALSE(ptx.find(".global .align 4 .f32 reg;") != std::string::npos);
  EXPECT_FALSE(ptx.find(".global .align 4 .f32 ret;") != std::string::npos);
  EXPECT_FALSE(ptx.find(".global .align 4 .f32 bra;") != std::string::npos);
  int* d_data;
  CHECK_CUDART(hipMalloc((void**)&d_data, sizeof(int)));
  int h_data = 3;
  CHECK_CUDART(
      hipMemcpy(d_data, &h_data, sizeof(int), hipMemcpyHostToDevice));
  // TODO: Should redirect stdout to avoid the printf message in the test log.
  ASSERT_EQ(compiled->link()
                ->load()
                ->get_kernel("foo_kernel")
                ->configure(1, 1)
                ->launch(d_data),
            "");
  CHECK_CUDART(
      hipMemcpy(&h_data, d_data, sizeof(int), hipMemcpyDeviceToHost));
  EXPECT_EQ(h_data, 16);
  CHECK_CUDART(hipFree(d_data));
}

TEST(Jitify2Test, ArchFlags) {
  static const char* const source = R"(
const int arch = __CUDA_ARCH__ / 10;
)";
  int current_arch = get_current_device_arch();
  int arch;
  // Test default behavior (automatic architecture detection).
  PreprocessedProgram preprocessed =
      Program("arch_flags_program", source)->preprocess();
  CompiledProgram program = preprocessed->compile();
  // Expect virtual architecture (compile to PTX).
  ASSERT_GT(program->ptx().size(), 0);
  ASSERT_EQ(program->cubin().size(), 0);
  ASSERT_EQ(program->link()->load()->get_global_value("arch", &arch), "");
  EXPECT_EQ(arch, current_arch);

  // Test explicit virtual architecture (compile to PTX).
  // Note: PTX is forwards compatible.
  program = preprocessed->compile("", {}, {"-arch=compute_50"});
  ASSERT_GT(program->ptx().size(), 0);
  ASSERT_EQ(program->cubin().size(), 0);
  ASSERT_EQ(program->link()->load()->get_global_value("arch", &arch), "");
  EXPECT_EQ(arch, 50);

  auto expect_cubin_size_if_available = [](size_t cubin_size) {
    if (jitify2::nvrtc().GetCUBIN()) {
      EXPECT_GT(cubin_size, 0);
    } else {
      EXPECT_EQ(cubin_size, 0);
    }
  };

  // Test explicit real architecture (may compile directly to CUBIN).
  program = preprocessed->compile(
      "", {}, {"-arch", "sm_" + std::to_string(current_arch)});
  EXPECT_GT(program->ptx().size(), 0);
  expect_cubin_size_if_available(program->cubin().size());
  ASSERT_EQ(program->link()->load()->get_global_value("arch", &arch), "");
  EXPECT_EQ(arch, current_arch);

  // Test automatic virtual architecture (compile to PTX).
  program = preprocessed->compile("", {}, {"-arch", "compute_."});
  EXPECT_GT(program->ptx().size(), 0);
  EXPECT_EQ(program->cubin().size(), 0);
  ASSERT_EQ(program->link()->load()->get_global_value("arch", &arch), "");
  EXPECT_EQ(arch, current_arch);

  // Test automatic real architecture (may compile directly to CUBIN).
  program = preprocessed->compile("", {}, {"-arch=sm_."});
  EXPECT_GT(program->ptx().size(), 0);
  expect_cubin_size_if_available(program->cubin().size());
  ASSERT_EQ(program->link()->load()->get_global_value("arch", &arch), "");
  EXPECT_EQ(arch, current_arch);

  // Test that preprocessing and compilation use separate arch flags.
  program = Program("arch_flags_program", source)
                ->preprocess({"-arch=sm_50"})
                ->compile("", {}, {"-arch=sm_."});
  EXPECT_GT(program->ptx().size(), 0);
  expect_cubin_size_if_available(program->cubin().size());
  ASSERT_EQ(program->link()->load()->get_global_value("arch", &arch), "");
  EXPECT_EQ(arch, current_arch);

  // Test that multiple architectures can be specified for preprocessing.
  program = Program("arch_flags_program", source)
                ->preprocess({"-arch=compute_50", "-arch=compute_52",
                              "-arch=compute_61"})
                ->compile("", {}, {"-arch=compute_."});
  EXPECT_GT(program->ptx().size(), 0);
  EXPECT_EQ(program->cubin().size(), 0);
  ASSERT_EQ(get_error(program), "");

  // Test that certain compiler options are automatically passed to the linker.
  LinkedProgram linked =
      Program("arch_flags_program", source)
          ->preprocess({"-maxrregcount=100", "-lineinfo", "-G"})
          ->compile()
          ->link();
  ASSERT_EQ(get_error(linked), "");
  std::unordered_multiset<std::string> linker_options(
      linked->linker_options().begin(), linked->linker_options().end());
  EXPECT_EQ(linker_options.count("-maxrregcount=100"), 1);
  EXPECT_EQ(linker_options.count("--generate-line-info"), 1);
  EXPECT_EQ(linker_options.count("-G"), 1);

  // Test with different option formats.
  linked = Program("arch_flags_program", source)
               ->preprocess({"--maxrregcount", "100", "--generate-line-info",
                             "--device-debug"})
               ->compile()
               ->link();
  ASSERT_EQ(get_error(linked), "");
  linker_options.clear();
  linker_options.insert(linked->linker_options().begin(),
                        linked->linker_options().end());
  EXPECT_EQ(linker_options.count("--maxrregcount=100"), 1);
  EXPECT_EQ(linker_options.count("--generate-line-info"), 1);
  EXPECT_EQ(linker_options.count("--device-debug"), 1);
}

struct Base {
  virtual ~Base() {}
};
template <typename T>
struct Derived : public Base {};

TEST(Jitify2Test, Reflection) {
  static const char* const source = R"(
struct Base { virtual ~Base() {} };
template <typename T>
struct Derived : public Base {};
template <typename T>
__global__ void type_kernel() {}
template <unsigned short N>
__global__ void nontype_kernel() {}
)";

  PreprocessedProgram preprog =
      Program("reflection_program", source)->preprocess();

  Template type_kernel("type_kernel");

#define JITIFY_TYPE_REFLECTION_TEST(T)                                   \
  EXPECT_EQ(                                                             \
      preprog->get_kernel(type_kernel.instantiate<T>())->lowered_name(), \
      preprog->get_kernel(type_kernel.instantiate({#T}))->lowered_name())

  JITIFY_TYPE_REFLECTION_TEST(const volatile float);
  JITIFY_TYPE_REFLECTION_TEST(const volatile float*);
  JITIFY_TYPE_REFLECTION_TEST(const volatile float&);
  JITIFY_TYPE_REFLECTION_TEST(Base * (const volatile float));
  JITIFY_TYPE_REFLECTION_TEST(const volatile float[4]);

#undef JITIFY_TYPE_REFLECTION_TEST

  typedef Derived<float> derived_type;
  const Base& base = derived_type();
  EXPECT_EQ(preprog->get_kernel(type_kernel.instantiate(instance_of(base)))
                ->lowered_name(),
            preprog->get_kernel(type_kernel.instantiate<derived_type>())
                ->lowered_name());

  Template nontype_kernel("nontype_kernel");

#define JITIFY_NONTYPE_REFLECTION_TEST(N)                                 \
  EXPECT_EQ(                                                              \
      preprog->get_kernel(nontype_kernel.instantiate(N))->lowered_name(), \
      preprog->get_kernel(nontype_kernel.instantiate({#N}))->lowered_name())

  JITIFY_NONTYPE_REFLECTION_TEST(7);
  JITIFY_NONTYPE_REFLECTION_TEST('J');

#undef JITIFY_NONTYPE_REFLECTION_TEST
}

TEST(Jitify2Test, BuiltinNumericLimitsHeader) {
  static const char* const source = R"(
#include <limits>
struct MyType {};
namespace std {
template<> class numeric_limits<MyType> {
 public:
  static MyType min() { return {}; }
  static MyType max() { return {}; }
};
}  // namespace std
template <typename T>
__global__ void my_kernel(T* data) {
  data[0] = std::numeric_limits<T>::min();
  data[1] = std::numeric_limits<T>::max();
}
)";
  PreprocessedProgram preprog =
      Program("builtin_numeric_limits_program", source)->preprocess();
  for (const auto& type :
       {"float", "double", "char", "signed char", "unsigned char", "short",
        "unsigned short", "int", "unsigned int", "long", "unsigned long",
        "long long", "unsigned long long", "MyType"}) {
    std::string kernel_inst = Template("my_kernel").instantiate(type);
    Kernel kernel =
        preprog->compile(kernel_inst)->link()->load()->get_kernel(kernel_inst);
    (void)kernel;
  }
}

TEST(Jitify2Test, CuRandKernel) {
  static const char* const source = R"(
#include <hiprand/hiprand_kernel.h>
__global__ void my_kernel() {}
)";
  Kernel kernel =
      Program("curand_program", source)
          // Note: --remove-unused-globals is added to remove huge precomputed
          // arrays that come from CURAND.
          ->preprocess({"-I" CUDA_INC_DIR, "--remove-unused-globals"})
          ->get_kernel("my_kernel");
  // TODO: Expand this test to actually call hiprand kernels and check outputs.
  (void)kernel;
}

TEST(Jitify2Test, Thrust) {
  // TODO: The need to include cstddef here under CUDA 12.0 may be related to
  //         the local/system include ambiguity problem in Jitify.
  // clang-format off
  static const char* const source = R"(
#include <cuda/std/cstddef>  // WAR for CUDA 12.0 build
#include <thrust/iterator/counting_iterator.h>
__global__ void my_kernel(thrust::counting_iterator<int> begin,
                          thrust::counting_iterator<int> end) {
})";
  // clang-format on
  // Checks that basic Thrust headers can be compiled.
#if CUDA_VERSION < 11000
  const char* cppstd = "-std=c++03";
#else
  const char* cppstd = "-std=c++14";
#endif
  PreprocessedProgram preprog = Program("thrust_program", source)
                                    ->preprocess({"-I" CUDA_INC_DIR, cppstd});
  ASSERT_EQ(get_error(preprog), "");
  ASSERT_EQ(get_error(preprog->compile()), "");
}

#if CUDA_VERSION >= 11000
TEST(Jitify2Test, CubBlockPrimitives) {
  static const char* const cub_program_source = R"(
// WAR for issue in CUB shipped with CUDA 11.4
// (https://github.com/NVIDIA/cub/issues/334)
// Note: We can't easily work around this inside Jitify itself.
// TODO(benbarsdell): Check exactly when this issue is fixed in CUB (<1.15.0?).
#include <cub/version.cuh>
#if CUB_VERSION >= 101200 && CUB_VERSION < 101500
#define ProcessFloatMinusZero BaseDigitExtractor<KeyT>::ProcessFloatMinusZero
#endif

#include <cub/block/block_load.cuh>
#include <cub/block/block_radix_sort.cuh>
#include <cub/block/block_reduce.cuh>
#include <cub/block/block_store.cuh>

template <int BLOCK_SIZE, int PER_THREAD>
__global__ void my_kernel(float* data) {
  typedef hipcub::BlockLoad<float, BLOCK_SIZE, PER_THREAD,
                         hipcub::BLOCK_LOAD_VECTORIZE>
      BlockLoad;
  typedef hipcub::BlockRadixSort<float, BLOCK_SIZE, PER_THREAD> BlockSort;
  typedef hipcub::BlockReduce<float, BLOCK_SIZE> BlockReduce;
  typedef hipcub::BlockStore<float, BLOCK_SIZE, PER_THREAD,
                          hipcub::BLOCK_STORE_VECTORIZE>
      BlockStore;
  __shared__ union {
    typename BlockLoad::TempStorage load;
    typename BlockSort::TempStorage sort;
    typename BlockReduce::TempStorage reduce;
    typename BlockStore::TempStorage store;
    float sum;
  } temp_storage;
  float thread_data[PER_THREAD];
  BlockLoad(temp_storage.load).Load(data, thread_data);
  __syncthreads();
  BlockSort(temp_storage.sort).Sort(thread_data);
  __syncthreads();
  float sum = BlockReduce(temp_storage.reduce).Sum(thread_data);
  __syncthreads();
  if (threadIdx.x == 0) {
    temp_storage.sum = sum;
  }
  __syncthreads();
  sum = temp_storage.sum;
#pragma unroll
  for (int i = 0; i < PER_THREAD; ++i) {
    thread_data[i] *= 1.f / sum;
  }
  __syncthreads();
  BlockStore(temp_storage.store).Store(data, thread_data);
}
)";
  int block_size = 64;
  int per_thread = 4;
  int n = block_size * per_thread;
  std::vector<float> h_data(n);
  float sum = 0;
  for (int i = 0; i < n; ++i) {
    // Start with values sorted in reverse.
    h_data[i] = (float)(n - 1 - i);
    sum += h_data[i];
  }
  // Shuffle the values a bit.
  std::swap(h_data[3], h_data[7]);
  std::swap(h_data[10], h_data[20]);
  std::vector<float> h_expected(n);
  for (int i = 0; i < n; ++i) {
    // Expected sorted and normalized.
    h_expected[i] = (float)i / sum;
  }
  std::vector<float> h_result(n);
  float* d_data;
  CHECK_CUDART(hipMalloc((void**)&d_data, n * sizeof(float)));
  CHECK_CUDART(hipMemcpy(d_data, h_data.data(), n * sizeof(float),
                          hipMemcpyHostToDevice));

  std::string kernel_inst =
      Template("my_kernel").instantiate(block_size, per_thread);
  Kernel kernel = Program("cub_program", cub_program_source)
                      ->preprocess({"-I" CUB_DIR, "-I" CUDA_INC_DIR})
                      ->compile(kernel_inst)
                      ->link()
                      ->load()
                      ->get_kernel(kernel_inst);
  kernel->configure(1, block_size)->launch(d_data);

  CHECK_CUDART(hipMemcpy(h_result.data(), d_data, n * sizeof(float),
                          hipMemcpyDeviceToHost));
  for (int i = 0; i < n; ++i) {
    EXPECT_FLOAT_EQ(h_result[i], h_expected[i]);
  }
  CHECK_CUDART(hipFree(d_data));
}
#endif  // CUDA_VERSION >= 11000

#if CUDA_VERSION >= 11000
TEST(Jitify2Test, LibCudaCxx) {
  // Test that each libcudacxx header can be compiled on its own.
  for (const std::string header :
       {"atomic", "barrier", "cassert", "cfloat", "chrono", "climits",
        "cstddef", "cstdint", "ctime", "functional", "latch",
        /*"limits",*/ "ratio", "semaphore", "type_traits", "utility"}) {
    std::string source =
        "#include <cuda/std/" + header + ">\n__global__ void my_kernel() {}";
    // Note: The -arch flag here is required because "CUDA atomics are
    // only supported for sm_60 and up on *nix and sm_70 and up on
    // Windows."
    Program("libcudacxx_program", source)
        ->preprocess({"-I" CUDA_INC_DIR, "-arch=compute_70",
                      "-no-builtin-headers", "-no-preinclude-workarounds",
                      "-no-system-headers-workaround",
                      "-no-replace-pragma-once"})
        ->get_kernel("my_kernel");
  }
  // WAR for bug in cuda/std/limits that is missing include cuda/std/climits.
  static const char* const source = R"(
#include <cuda/std/climits>
#include <cuda/std/limits>
__global__ void my_kernel() {}
)";
  Program("libcudacxx_program", source)
      ->preprocess({"-I" CUDA_INC_DIR, "-arch=compute_70",
                    "-no-builtin-headers", "-no-preinclude-workarounds",
                    "-no-system-headers-workaround", "-no-replace-pragma-once"})
      ->get_kernel("my_kernel");
}
#endif  // CUDA_VERSION >= 11000

TEST(Jitify2Test, AssertHeader) {
  static const char* const source = R"(
#include <cassert>
__global__ void my_assert_kernel() {
  assert(0 == 1);
}
)";
  // TODO: Should temporarily redirect stderr while executing this kernel and
  // check that the assertion message is printed (this will also avoid printing
  // the assertion message to the test log).
  // Checks that cassert works as expected.
  Program("assert_program", source)
      ->preprocess()
      ->get_kernel("my_assert_kernel")
      ->configure(1, 1)
      ->launch();
  ASSERT_EQ(hipDeviceSynchronize(), hipErrorAssert);
  // NOTE: Assertion failure is a sticky error in CUDA, so the process can no
  // longer be used for CUDA operations after this point.
}

TEST(Jitify2Test, Minify) {
  static const char* const name = "my_program";
  // This source is intentionally tricky to parse so that it stresses the
  // minification algorithm.
  static const std::string source = R"(
//#define FOO foo
//#define BAR(call)                             \
//  do {                                        \
//    call;                                     \
//  } while (0)

#ifndef __CUDACC_RTC__
    #define FOOBAR
    #define BARFOO
#else
    #define MY_CHAR_BIT 8
    #define __MY_CHAR_UNSIGNED__ ('\xff' > 0) // CURSED
    #if __MY_CHAR_UNSIGNED__
        #define MY_CHAR_MIN 0
        #define MY_CHAR_MAX UCHAR_MAX
    #else
        #define MY_CHAR_MIN SCHAR_MIN
        #define MY_CHAR_MAX SCHAR_MAX
    #endif
#endif
/*
This will
all be
"trickily"
removed
hopefully.*/

const char* const foo = R"foo(abc\def
ghi"')foo";  // )'

  #include <iterator>  // Here's a comment
  #include <tuple>  // Here's another comment

const char* const linecont_str = "line1 \
line2";
const char c = '\xff';

#include <hip/hip_runtime.h>
#if CUDA_VERSION >= 11000
// CUB headers can be tricky to parse.
#include <cub/block/block_load.cuh>
#include <cub/block/block_radix_sort.cuh>
#include <cub/block/block_reduce.cuh>
#include <cub/block/block_store.cuh>
#endif  // CUDA_VERSION >= 11000

#include "example_headers/my_header1.cuh"
__global__ void my_kernel() {}
)";
  PreprocessedProgram preprog =
      Program(name, source)->preprocess({"-I" CUB_DIR, "-I" CUDA_INC_DIR});
  ASSERT_EQ(get_error(preprog), "");
  CompiledProgram compiled = preprog->compile();
  ASSERT_EQ(get_error(compiled), "");
  std::string orig_ptx = compiled->ptx();

  preprog = Program(name, source)
                ->preprocess({"-I" CUB_DIR, "-I" CUDA_INC_DIR, "--minify"});
  ASSERT_EQ(get_error(preprog), "");
  EXPECT_LT(preprog->source().size(), source.size());
  compiled = preprog->compile();
  ASSERT_EQ(get_error(compiled), "");
  ASSERT_EQ(compiled->ptx(), orig_ptx);
}

int main(int argc, char** argv) {
  hipSetDevice(0);
  // Initialize the driver context (avoids "initialization error"/"context is
  // destroyed").
  hipFree(0);
  ::testing::InitGoogleTest(&argc, argv);
  // Test order is actually undefined, so we use filters to force the
  // AssertHeader test to run last.
  ::testing::GTEST_FLAG(filter) += ":-Jitify2Test.AssertHeader";
  int result = RUN_ALL_TESTS();
  ::testing::GTEST_FLAG(filter) = "Jitify2Test.AssertHeader";
  return result | RUN_ALL_TESTS();
}
